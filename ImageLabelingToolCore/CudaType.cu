#include "hip/hip_runtime.h"
#include "CudaType.cuh"
#include <iostream>
#include <hip/hip_runtime_api.h>

namespace ImageLabelTool
{
	namespace Core
	{
		namespace Cuda
		{
			using namespace std;

			template struct CudaVolume<unsigned char>;
			template struct CudaVolume<unsigned short>;
			template struct CudaVolume<float>;
			template <typename PixelType> CudaVolume<PixelType>::~CudaVolume() {
				Free();
			}
			template <typename PixelType> int CudaVolume<PixelType>::GetDeviceID() {
				return m_deviceId;
			}
			template <typename PixelType> PixelType* CudaVolume<PixelType>::GetDevicePtr() {
				return *(PixelType**)m_devicePtr;
			}
			template <typename PixelType> dim3 CudaVolume<PixelType>::GetSize() {
				return m_size;
			}
			template <typename PixelType> float3 CudaVolume<PixelType>::GetSpacing() {
				return m_spacing;
			}
			template <typename PixelType> float3 CudaVolume<PixelType>::GetOrigin() {
				return m_origin;
			}
			template <typename PixelType> ulonglong3 CudaVolume<PixelType>::GetOffset() {
				return m_offset;
			}
			template <typename PixelType> ulonglong3 CudaVolume<PixelType>::GetByteSize() {
				return m_byteSize;
			}
			template <typename PixelType> void CudaVolume<PixelType>::SetSpacing(float3 spacing) {
				m_spacing = spacing;
			}
			template <typename PixelType> void CudaVolume<PixelType>::SetOrigin(float3 origin) {
				m_origin = origin;
			}
			template <typename PixelType> void CudaVolume<PixelType>::Import(int deviceID, void* d_ptr, dim3 size) {
				Free();
				m_deviceId = deviceID;
				m_isImported = true;
				m_devicePtr = (PixelType*)d_ptr;
				m_size = size;
				m_offset.x = size.x;
				m_offset.y = m_offset.x * size.y;
				m_offset.z = m_offset.y * size.z;
				m_byteSize.x = m_offset.x * sizeof(PixelType);
				m_byteSize.y = m_offset.y * sizeof(PixelType);
				m_byteSize.z = m_offset.z * sizeof(PixelType);
			}
			template <typename PixelType> void CudaVolume<PixelType>::Allocate(int deviceID, dim3 size) {
				Free();
				m_size = size;
				m_offset.x = size.x;
				m_offset.y = m_offset.x * size.y;
				m_offset.z = m_offset.y * size.z;
				m_byteSize.x = m_offset.x * sizeof(PixelType);
				m_byteSize.y = m_offset.y * sizeof(PixelType);
				m_byteSize.z = m_offset.z * sizeof(PixelType);
				CHECK_CUDA_ERROR(hipSetDevice(deviceID));
				CHECK_CUDA_ERROR(hipMalloc((void**)&m_devicePtr, m_byteSize.z));
				m_deviceId = deviceID;
				m_isImported = false;
			}
			template <typename PixelType> void CudaVolume<PixelType>::Free() {
				if (!m_isImported && m_devicePtr)
					CHECK_CUDA_ERROR(hipFree(m_devicePtr));
				m_devicePtr = NULL;
			}
		}
	}
}
